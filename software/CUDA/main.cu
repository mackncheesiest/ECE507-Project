#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Apr 23, 2019
 *      Author: josh
 */

#include <string>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <stdlib.h>

#include "data.h"

#define NUM_ITER 360000
#define DESTROY_PROB 0.00
#define PRINT_ITER 1000
#define SEED_VAL 1234

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t errCode, const char *file, int line, bool abort=true) {
  if (errCode != hipSuccess) {
    fprintf(stderr, "GPU Assertion: %s %s %d\n", hipGetErrorString(errCode), file, line);
    if (abort) exit(errCode);
  }
}

#define KERNEL_VERSION 4

#if KERNEL_VERSION==1
__global__ void spinUpdateKernel(int* spinArr, int* spinArr_temp,
                                 const int* __restrict__ interactionArrHorizontal, const int* __restrict__ interactionArrVertical) {

  int IS0, IS1, currRow, currCol, a, b, newSpin;

  hiprandState rand_state;
  hiprand_init((blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x),
              (blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x), 0, &rand_state);

  for (unsigned int i = 0; i < (GRID_LEN-1)/blockDim.y + 1; i++) {
    for (unsigned int j = 0; j < (GRID_LEN-1)/blockDim.x + 1; j++) {
      currRow = i * blockDim.y + threadIdx.y;
      currCol = j * blockDim.x + threadIdx.x;
      if (currRow < GRID_LEN && currCol < GRID_LEN) {
        /*
        IS0 = magneticFieldArr0[currRow * GRID_LEN + currCol];
        IS1 = magneticFieldArr1[currRow * GRID_LEN + currCol];
        */

        a = 0;
        b = 0;

        //Compute influence of external magnetic field
        /*
        if (IS0 == 1) {
          if (IS1 == 1) {
            a++;
          }
          else {
            b++;
          }
        }
        */

        // Compute influence of left neighbor
        if (currCol > 0) {
          if (spinArr[currRow * GRID_LEN + currCol - 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol - 1]) a++;
          else b++;
        }
        // Compute influence of right neighbor
        if (currCol < GRID_LEN-2) {
          if (spinArr[currRow * GRID_LEN + currCol + 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol + 1]) a++;
          else b++;
        }
        // Compute influence of up neighbor
        if (currRow > 0) {
          if (spinArr[(currRow-1) * GRID_LEN + currCol] == interactionArrVertical[(currRow-1) * (GRID_LEN-1) + currCol]) a++;
          else b++;
        }
        // Compute influence of down neighbor
        if (currRow < GRID_LEN-2) {
          if (spinArr[(currRow+1) * GRID_LEN + currCol] == interactionArrVertical[(currRow+1) * (GRID_LEN-1) + currCol]) a++;
          else b++;
        }

        // Update state
        if (a > b) {
          newSpin = 1;
        } else if (a < b) {
          newSpin = -1;
        } else {
          newSpin = hiprand_uniform(&rand_state) <= 0.5 ? -1 : 1;
//          newSpin = ((a * b) % 2) == 1 ? 1 : -1;
        }

        spinArr_temp[currRow * GRID_LEN + currCol] = newSpin;

        // Randomly flip state
        /*
        if (destroySpinState(DESTROY_PROB)) {
          spinArr_temp[currRow * GRID_LEN + currCol] = -spinArr_temp[currRow * GRID_LEN + currCol];
        }
        */
      }
    }
  }
}
#elif KERNEL_VERSION==2

#define TILE_DIM 16
#define MASK_WIDTH 3
#define MASK_RADIUS MASK_WIDTH/2
#define PADDED_DIM (TILE_DIM + MASK_WIDTH - 1)

__global__ void spinUpdateKernel(int* spinArr, int* spinArr_temp,
                                 const int* __restrict__ interactionArrHorizontal, const int* __restrict__ interactionArrVertical) {

  int IS0, IS1, currRow, currCol, a, b, newSpin;

  hiprandState rand_state;
  hiprand_init(clock64(), (blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x), 0, &rand_state);

  __shared__ float shared_spinArr[PADDED_DIM * PADDED_DIM];
  __shared__ float shared_interactionArrHorizontal[PADDED_DIM * (PADDED_DIM-1)];
  __shared__ float shared_interactionArrVertical[(PADDED_DIM-1) * PADDED_DIM];

  // Load Batch 1 (Spin Arr):
  int dest = threadIdx.y * TILE_DIM + threadIdx.x;
  int destY = dest / PADDED_DIM;
  int destX = dest % PADDED_DIM;
  int srcY = blockIdx.y * TILE_DIM + destY - MASK_RADIUS;
  int srcX = blockIdx.x * TILE_DIM + destX - MASK_RADIUS;
  int src = srcY * GRID_LEN + srcX;

  if ((srcY >= 0) && (srcX >= 0)) {
    if (srcY < GRID_LEN && srcX < GRID_LEN) {
      shared_spinArr[destY * PADDED_DIM + destX] = spinArr[src];
    } else {
      shared_spinArr[destY * PADDED_DIM + destX] = 0;
    }

    if ((srcY < GRID_LEN) && (srcX < GRID_LEN-1)) {
      shared_interactionArrHorizontal[destY * PADDED_DIM + destX] = interactionArrHorizontal[src];
    } else {
      shared_interactionArrHorizontal[destY * PADDED_DIM + destX] = 0;
    }

    if ((srcY < GRID_LEN-1) && (srcX < GRID_LEN)) {
      shared_interactionArrVertical[destY * PADDED_DIM + destX] = interactionArrVertical[src];
    } else {
      shared_interactionArrVertical[destY * PADDED_DIM + destX] = 0;
    }
  }

  // Load Batch 2 (Spin Arr):
  // Note: this assumes that you have inner threads >= outer threads,
  dest = threadIdx.y * TILE_DIM + threadIdx.x + TILE_DIM * TILE_DIM;
  destY = dest / PADDED_DIM;
  destX = dest % PADDED_DIM;
  srcY = blockIdx.y * TILE_DIM + destY - MASK_RADIUS;
  srcX = blockIdx.x * TILE_DIM + destX - MASK_RADIUS;
  src = srcY * GRID_LEN + srcX;

  if (destY < PADDED_DIM) {
    if ((srcY >= 0) && (srcX >= 0)) {
      if (srcY < GRID_LEN && srcX < GRID_LEN) {
        shared_spinArr[destY * PADDED_DIM + destX] = spinArr[src];
      } else {
        shared_spinArr[destY * PADDED_DIM + destX] = 0;
      }

      if ((srcY < GRID_LEN) && (srcX < GRID_LEN-1)) {
        shared_interactionArrHorizontal[destY * PADDED_DIM + destX] = interactionArrHorizontal[src];
      } else {
        shared_interactionArrHorizontal[destY * PADDED_DIM + destX] = 0;
      }

      if ((srcY < GRID_LEN-1) && (srcX < GRID_LEN)) {
        shared_interactionArrVertical[destY * PADDED_DIM + destX] = interactionArrVertical[src];
      } else {
        shared_interactionArrVertical[destY * PADDED_DIM + destX] = 0;
      }
    }
  }

  __syncthreads();

  currRow = blockIdx.y * TILE_DIM + threadIdx.y;
  currCol = blockIdx.x * TILE_DIM + threadIdx.x;
  if (currRow < GRID_LEN && currCol < GRID_LEN) {
    /*
    IS0 = magneticFieldArr0[currRow * GRID_LEN + currCol];
    IS1 = magneticFieldArr1[currRow * GRID_LEN + currCol];
    */

    a = 0;
    b = 0;

    //Compute influence of external magnetic field
    /*
    if (IS0 == 1) {
      if (IS1 == 1) {
        a++;
      }
      else {
        b++;
      }
    }
    */

    // Compute influence of left neighbor
    if (currCol > 0) {
      if (shared_spinArr[currRow * PADDED_DIM + currCol - 1] == shared_interactionArrHorizontal[currRow * PADDED_DIM + currCol - 1]) a++;
      else b++;
    }
    // Compute influence of right neighbor
    if (currCol < GRID_LEN-2) {
      if (shared_spinArr[currRow * PADDED_DIM + currCol + 1] == shared_interactionArrHorizontal[currRow * PADDED_DIM + currCol + 1]) a++;
      else b++;
    }
    // Compute influence of up neighbor
    if (currRow > 0) {
      if (shared_spinArr[(currRow-1) * PADDED_DIM + currCol] == shared_interactionArrVertical[(currRow-1) * (PADDED_DIM-1) + currCol]) a++;
      else b++;
    }
    // Compute influence of down neighbor
    if (currRow < GRID_LEN-2) {
      if (shared_spinArr[(currRow+1) * PADDED_DIM + currCol] == shared_interactionArrVertical[(currRow+1) * (PADDED_DIM-1) + currCol]) a++;
      else b++;
    }

    // Update state
    if (a > b) {
      newSpin = 1;
    } else if (a < b) {
      newSpin = -1;
    } else {
      newSpin = hiprand_uniform(&rand_state) <= 0.5 ? -1 : 1;
    }

    spinArr_temp[currRow * GRID_LEN + currCol] = newSpin;

    // Randomly flip state
    /*
    if (destroySpinState(DESTROY_PROB)) {
      spinArr_temp[currRow * GRID_LEN + currCol] = -spinArr_temp[currRow * GRID_LEN + currCol];
    }
    */
  }
}
#elif KERNEL_VERSION==3

/*
 * Assumes that blockDim.x == GRID_LEN (and implicitly that GRID_LEN is smaller than maximum x block size)
 *              blockDim.y == (1024-1)/GRID_LEN + 1
 */

__global__ void spinUpdateKernel(int* spinArr, int* spinArr_temp,
                                 const int* __restrict__ interactionArrHorizontal, const int* __restrict__ interactionArrVertical) {

  int IS0, IS1, globalRow, globalCol, currRow, currCol, a, b, newSpin;

  globalRow = blockIdx.y + threadIdx.y;
  globalCol = threadIdx.x;

  hiprandState rand_state;
//  hiprand_init(clock64(), (blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x), 0, &rand_state);
  hiprand_init((blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x),
              (blockDim.y * blockIdx.y + threadIdx.y) * (blockDim.x * blockIdx.x + threadIdx.x), 0, &rand_state);

  __shared__ float shared_spinArr[((GRID_LEN-1)/1024 + 1 + 2) * GRID_LEN];
//  __shared__ float shared_interactionArrHorizontal[blockDim.y * (GRID_LEN-1)];
//  __shared__ float shared_interactionArrVertical[(blockDim.y-1) * GRID_LEN];

  for (int offset = -1; offset < (blockDim.y+2); offset += blockDim.y) {
    shared_spinArr[(offset+threadIdx.y)] = spinArr[((offset+blockIdx.y) * blockDim.y + threadIdx.y) * GRID_LEN + blockIdx.x];
  }

  __syncthreads();

  currRow = threadIdx.y;
  currCol = threadIdx.x;
  if (globalRow < GRID_LEN && globalCol < GRID_LEN) {
    /*
    IS0 = magneticFieldArr0[currRow * GRID_LEN + currCol];
    IS1 = magneticFieldArr1[currRow * GRID_LEN + currCol];
    */

    a = 0;
    b = 0;

    //Compute influence of external magnetic field
    /*
    if (IS0 == 1) {
      if (IS1 == 1) {
        a++;
      }
      else {
        b++;
      }
    }
    */

    // Compute influence of left neighbor
    if (globalCol > 0) {
      if (shared_spinArr[currRow * GRID_LEN + currCol - 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol - 1]) a++;
      else b++;
    }
    // Compute influence of right neighbor
    if (globalCol < GRID_LEN-2) {
      if (shared_spinArr[currRow * GRID_LEN + currCol + 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol + 1]) a++;
      else b++;
    }
    // Compute influence of up neighbor
    if (globalRow > 0) {
      if (shared_spinArr[(currRow-1) * GRID_LEN + currCol] == interactionArrVertical[(currRow-1) * (GRID_LEN-1) + currCol]) a++;
      else b++;
    }
    // Compute influence of down neighbor
    if (globalRow < GRID_LEN-2) {
      if (shared_spinArr[(currRow+1) * GRID_LEN + currCol] == interactionArrVertical[(currRow+1) * (GRID_LEN-1) + currCol]) a++;
      else b++;
    }

    // Update state
    if (a > b) {
      newSpin = 1;
    } else if (a < b) {
      newSpin = -1;
    } else {
      newSpin = hiprand_uniform(&rand_state) <= 0.5 ? -1 : 1;
    }

    spinArr_temp[globalRow * GRID_LEN + globalCol] = newSpin;

    // Randomly flip state
    /*
    if (destroySpinState(DESTROY_PROB)) {
      spinArr_temp[currRow * GRID_LEN + currCol] = -spinArr_temp[currRow * GRID_LEN + currCol];
    }
    */
  }
}
#elif KERNEL_VERSION==4
__global__ void spinUpdateKernel(int* spinArr, int* spinArr_temp,
                                 const int* __restrict__ interactionArrHorizontal, const int* __restrict__ interactionArrVertical,
                                 const unsigned int* __restrict__ randomSpins, const int randOffset) {

  int IS0, IS1, currRow, currCol, a, b, newSpin;

  for (unsigned int i = 0; i < (GRID_LEN-1)/blockDim.y + 1; i++) {
    for (unsigned int j = 0; j < (GRID_LEN-1)/blockDim.x + 1; j++) {
      currRow = i * blockDim.y + threadIdx.y;
      currCol = j * blockDim.x + threadIdx.x;
      if (currRow < GRID_LEN && currCol < GRID_LEN) {
        /*
        IS0 = magneticFieldArr0[currRow * GRID_LEN + currCol];
        IS1 = magneticFieldArr1[currRow * GRID_LEN + currCol];
        */

        a = 0;
        b = 0;

        //Compute influence of external magnetic field
        /*
        if (IS0 == 1) {
          if (IS1 == 1) {
            a++;
          }
          else {
            b++;
          }
        }
        */

        // Compute influence of left neighbor
        if (currCol > 0) {
          if (spinArr[currRow * GRID_LEN + currCol - 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol - 1]) a++;
          else b++;
        }
        // Compute influence of right neighbor
        if (currCol < GRID_LEN-2) {
          if (spinArr[currRow * GRID_LEN + currCol + 1] == interactionArrHorizontal[currRow * GRID_LEN + currCol + 1]) a++;
          else b++;
        }
        // Compute influence of up neighbor
        if (currRow > 0) {
          if (spinArr[(currRow-1) * GRID_LEN + currCol] == interactionArrVertical[(currRow-1) * (GRID_LEN-1) + currCol]) a++;
          else b++;
        }
        // Compute influence of down neighbor
        if (currRow < GRID_LEN-2) {
          if (spinArr[(currRow+1) * GRID_LEN + currCol] == interactionArrVertical[(currRow+1) * (GRID_LEN-1) + currCol]) a++;
          else b++;
        }

        // Update state
        if (a > b) {
          newSpin = 1;
        } else if (a < b) {
          newSpin = -1;
        } else {
          newSpin = randomSpins[currRow * GRID_LEN + currCol + randOffset] % 2 == 0 ? -1 : 1;
//          newSpin = ((a * b) % 2) == 1 ? 1 : -1;
        }

        spinArr_temp[currRow * GRID_LEN + currCol] = newSpin;

        // Randomly flip state
        /*
        if (destroySpinState(DESTROY_PROB)) {
          spinArr_temp[currRow * GRID_LEN + currCol] = -spinArr_temp[currRow * GRID_LEN + currCol];
        }
        */
      }
    }
  }
}
#endif

void writeToFile(int* spinArr, std::string filename) {
     // Write results to file
    FILE *fp;
    fp = fopen(filename.c_str(), "w");

    if (fp == NULL) {
        printf("Unable to create output file.\n");
        exit(EXIT_FAILURE);
    }

    for (int row = 0; row < GRID_LEN; row++) {
        for (int col = 0; col < GRID_LEN; col++) {
            fprintf(fp, "%d ", spinArr[row * GRID_LEN + col]);
        }
        fprintf(fp, "\n");
    }

    fclose(fp);
}

int main(void) {

  int *h_outputSpinArr = (int*) malloc(GRID_LEN * GRID_LEN * sizeof(int));
  int *d_spinArr_1, *d_spinArr_2, *d_spinArr_swap, *d_magneticFieldArr0, *d_magneticFieldArr1, *d_interactionArrHorizontal, *d_interactionArrVertical;

  unsigned int *d_randomSpins, randOffset;

  gpuErrchk(hipMalloc((void**) &d_spinArr_1, GRID_LEN * GRID_LEN * sizeof(int)));
  gpuErrchk(hipMalloc((void**) &d_spinArr_2, GRID_LEN * GRID_LEN * sizeof(int)));
  gpuErrchk(hipMalloc((void**) &d_magneticFieldArr0, GRID_LEN * GRID_LEN * sizeof(int)));
  gpuErrchk(hipMalloc((void**) &d_magneticFieldArr1, GRID_LEN * GRID_LEN * sizeof(int)));
  gpuErrchk(hipMalloc((void**) &d_interactionArrHorizontal, GRID_LEN * (GRID_LEN-1) * sizeof(int)));
  gpuErrchk(hipMalloc((void**) &d_interactionArrVertical, (GRID_LEN-1) * GRID_LEN * sizeof(int)));

  gpuErrchk(hipMalloc((void**) &d_randomSpins, (GRID_LEN * GRID_LEN + 20) * sizeof(int)));

  gpuErrchk(hipMemcpy(d_spinArr_1, spinArr, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_magneticFieldArr0, magneticFieldArr0, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_magneticFieldArr1, magneticFieldArr1, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_interactionArrHorizontal, interactionArrHorizontal, GRID_LEN * (GRID_LEN-1) * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_interactionArrVertical, interactionArrVertical, (GRID_LEN-1) * GRID_LEN * sizeof(int), hipMemcpyHostToDevice));

  srand(SEED_VAL);

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, SEED_VAL);
  hiprandGenerate(gen, d_randomSpins, GRID_LEN * GRID_LEN + 20);

  #if KERNEL_VERSION==1
  dim3 dimBlock(min(GRID_LEN, 1024), max(1024/GRID_LEN, 1), 1);
  dim3 dimGrid(1, GRID_LEN/max(1024/GRID_LEN, 1), 1);
  #elif KERNEL_VERSION==2
  const int blockDim = min(GRID_LEN, 16);
  dim3 dimBlock(blockDim, blockDim);
  dim3 dimGrid((GRID_LEN-1)/blockDim + 1, (GRID_LEN-1)/blockDim + 1, 1);
  #elif KERNEL_VERSION==3
  dim3 dimBlock(GRID_LEN, max(1024/GRID_LEN, 1), 1);
  dim3 dimGrid(1, GRID_LEN/max(1024/GRID_LEN, 1), 1);
  #elif KERNEL_VERSION==4
  dim3 dimBlock(min(GRID_LEN, 1024), max(1024/GRID_LEN, 1), 1);
  dim3 dimGrid(1, GRID_LEN/max(1024/GRID_LEN, 1), 1);
  #endif

  for (int i = 0; i < NUM_ITER; i++) {
    randOffset = rand() % 20;
    #if KERNEL_VERSION==4
    spinUpdateKernel<<<dimBlock, dimGrid>>>(d_spinArr_1, d_spinArr_2, d_interactionArrHorizontal, d_interactionArrVertical, d_randomSpins, randOffset);
    #else
    spinUpdateKernel<<<dimBlock, dimGrid>>>(d_spinArr_1, d_spinArr_2, d_interactionArrHorizontal, d_interactionArrVertical);
    #endif
    gpuErrchk(hipPeekAtLastError());

    d_spinArr_swap = d_spinArr_1;
    d_spinArr_1 = d_spinArr_2;
    d_spinArr_2 = d_spinArr_swap;
    if (i % PRINT_ITER == 0) {
      printf("Finished enqueuing iteration %d...\n", i);
    }
  }

//  gpuErrchk(hipMemcpy(h_outputSpinArr, d_spinArr, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyDeviceToHost));
  printf("Done enqueuing kernels on host, waiting for GPU to finish...\n");
  hipDeviceSynchronize();
  printf("GPU finished! Copying output...\n");
  if (NUM_ITER % 2 == 0) {
    gpuErrchk(hipMemcpy(h_outputSpinArr, d_spinArr_1, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyDeviceToHost));
  } else {
    gpuErrchk(hipMemcpy(h_outputSpinArr, d_spinArr_2, GRID_LEN * GRID_LEN * sizeof(int), hipMemcpyDeviceToHost));
  }

  writeToFile(h_outputSpinArr, "final.txt");
  printf("Done!\n");

  return 0;
}

